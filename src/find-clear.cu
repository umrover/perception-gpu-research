#include "hip/hip_runtime.h"
#include "common.hpp"
#include <float.h>
//Find clear path
//Checks to see if minx and maxX are between the two projected lines
//It not checks to see if the line segment between the two minX and maxX intersect the projected line
//If such points are found launch write them to place in shared memory
//If such poitns are not found then in shared memory space write float_max or float_min
//SyncThreads and perform parallel reduction to find minX and maxX thread values
//launch left and right path kernels using new calculated lines
//Left and right path kernels will perform the same steps except in a particular direction
//Assumes we don't have over 1024 clusters

class compareLine {
public:
    int xIntercept;
    float slope;
    
    __device__ compareLine(float angle_in, int xInt_in) : xIntercept{xInt_in}, 
                        slope{tan(angle_in*PI/180)} {
                            if(slope != 0) {
                                slope = 1/slope;
                            }
                        }

    //Returns 1 if point is right of line, 0 if on, -1 if left of line
    __device__ int operator()(float x, float y) {
        
        //Make sure don't divide by 0
        float xc = xIntercept; //x calculated
        if(slope != 0) {
            xc = y/slope+xIntercept; //Find x value on line with same y value as input point
        }
            
        //Point is right of the line
        if(x > xc) {
            return 1;
        }
        //Point is on the line
        else if (x == xc) {
            return 0;
        }
        //Point is left of the line
        else {
            return -1;
        } 
    }

    //Assumes x1 < x2
    __device__ bool operator()(float x1, float y1, float x2, float y2) {
        if(x1 != x2){
            if(slope != 0){
                float slopeSeg = (y2-y1)/(x2-x1);
                float xIntersect = (-slopeSeg*x1+y1-xIntercept)/(slope-slopeSeg);
                return (xIntersect < x2 && xIntersect > x1);
            }
            //Check if left of line and right of line if slope is undefined
            else if(this->operator()(x1,y1) < 1 && this->operator()(x2,y2) > -1) return true; 
        }
        return false;
        
    }
};

int findNextLargestSquare(int num){
    int exp = log2(num) + 1;
    return pow(2,exp);
}

//Finds the leftmost or rightmost obstacle in a given path, calculates a new path based on where this obstacle is, 
//and then checks that path for obstacles. Direction of 1 is right and 0 is left
__global__ void findAngleOffCenterKernel(float* minXG, float* maxXG, float* minZG, int numClusters, float* bearing, int direction) {
    
    if(!(*bearing)) return;

    //Declare variables
    __shared__ float maxXS[MAX_THREADS];
    __shared__ float minXS[MAX_THREADS];
    __shared__ float minZS[MAX_THREADS];
    __shared__ bool obstacle;
    __shared__ float slope;

    if(threadIdx.x == 0) slope = 0;
    if(threadIdx.x >= numClusters) return;

    //Copy over data from global to local
    float maxX = maxXG[threadIdx.x];
    float minX = minXG[threadIdx.x];
    float minZ = minZG[threadIdx.x];

    *bearing = 0;

    do {

        if(threadIdx.x == 0) obstacle = false; //Assume path we're checking is clear until we detect obstacle

        __syncthreads();

        //Creates functors to compare points against
        compareLine rightLine(slope, HALF_ROVER); //Is there a way to make these objects shared?
        compareLine leftLine(slope, -HALF_ROVER);

        //Checks if either of the threads points is between the two lines
        if((leftLine(maxX, minZ) > -1 && rightLine(maxX, minZ) < 1) || //check if maxX is between right and left
        (leftLine(minX, minZ) > -1 && rightLine(minX, minZ) < 1) || //Check if minX is between right and left
        (leftLine(minX, minZ, maxX, minZ) || rightLine(minZ, minZ, maxX, minZ))) { //check if lines intersect line seg
            obstacle = true;
            maxXS[threadIdx.x] = maxX;
            minXS[threadIdx.x] = minX;
            minZS[threadIdx.x] = minZ;
            printf("Point %d found in path\n", threadIdx.x);
        }
        else {
            maxXS[threadIdx.x] = FLT_MIN;
            minXS[threadIdx.x] = FLT_MAX;
        }

        __syncthreads();

        //Zero thread finds the min and max. Could do parallel reduction here, 
        //but so few clusters probably not worth the effort
        if(threadIdx.x == 0 && obstacle){
            float minXFinal = FLT_MAX, minZMinXFinal = 0, maxXFinal = FLT_MIN, minZMaxXFinal = 0; 
            for(int i = 0; i < numClusters; ++i ){
                if(maxXS[i] > maxXFinal) {
                    maxXFinal= maxXS[i];
                    minZMaxXFinal = minZS[i];
                }
                if(minXS[i] < minXFinal) {
                    minXFinal = minXS[i];
                    minZMinXFinal = minZS[i];
                }
            }
            
            int buffer = 0;
            //Finding slpoe off center
            float oppSideRTri = direction ? maxXFinal : minXFinal;
            float adjSideRTri = direction ? minZMaxXFinal : minZMinXFinal;//Length of adjacent side of right triangle
            oppSideRTri += direction ? buffer+HALF_ROVER : -(buffer+HALF_ROVER); //Calculate length of opposite side of right triangle
            slope = atan(oppSideRTri/adjSideRTri)*180/PI;//arctan(opposite/adjacent)
            if(direction == 0 && threadIdx.x == 0)printf("Leftmost: (%.1f, %.1f) Bearing: %.1f\n", minXFinal, minZMinXFinal, slope);
            if(direction == 1 && threadIdx.x == 0)printf("Rightmost: (%.1f, %.1f) Bearing: %.1f\n", maxXFinal, minZMaxXFinal, slope);   
        }

    } while(obstacle && fabs(slope) < 70);

    if(threadIdx.x == 0) printf("Bearing: %.1f\n", slope);
    if(threadIdx.x == 0) *bearing = slope; //Write to global memory
    
}
__global__ void findClearPathKernel(float* minXG, float* maxXG, float* minZG, int numClusters, float* leftBearing, float* rightBearing) {
        
    //Declare variables
    if(threadIdx.x >= numClusters) return;

    compareLine rightLine(0,HALF_ROVER); //Is there a way to make these objects shared?
    compareLine leftLine(0,-HALF_ROVER);
    __shared__ bool obstacle;

    if(threadIdx.x == 0) obstacle = false;

    //Copy over data from global to local
    float maxX = maxXG[threadIdx.x];
    float minX = minXG[threadIdx.x];
    float minZ = minZG[threadIdx.x];

    if(threadIdx.x == 0) printf("Vars are copied\n");

    //Check where point is relative to line
    //Since the z values of the min and max x values aren't stored we are going
    //to assume that they are both located at the min z value of the cluster

    //Checks if either of the points is between the two lines
    if((leftLine(maxX, minZ) > -1 && rightLine(maxX, minZ) < 1) || //check if maxX is between right and left
       (leftLine(minX, minZ) > -1 && rightLine(minX, minZ) < 1) || //Check if minX is between right and left
       (leftLine(minX, minZ, maxX, minZ) || rightLine(minZ, minZ, maxX, minZ))) { //check if lines intersect line seg
        obstacle = true;
    }

    __syncthreads();

    //Iterate through points to find mins and maxes
    //No need for parallel reduction, since cluster size is relatively small
    
    if(threadIdx.x == 0){
        if(!obstacle) { //Path is clear
            *leftBearing = 0;
            *rightBearing = 0;
            printf("Center Path Clear\n");
        }
        else { //Path is blocked find left and right angles
            *leftBearing = -1;
            *rightBearing = -1;
        }
    }
    
}

void testClearPath() {

    int testClusterSize = 4;
    float* minXG;
    float* maxXG;
    float* minZG;
    float* leftBearing;
    float* rightBearing;

    hipMalloc(&minXG, sizeof(float)*testClusterSize);
    hipMalloc(&maxXG, sizeof(float)*testClusterSize);
    hipMalloc(&minZG, sizeof(float)*testClusterSize);
    hipMalloc(&leftBearing, sizeof(float));
    hipMalloc(&rightBearing, sizeof(float));

    float minXCPU[testClusterSize] = { -6, -10, 6, -100, };
    float maxXCPU[testClusterSize] = { 6, -7, 7, -9};
    float minZCPU[testClusterSize] = { 10, 10, 10, 10};

    hipMemcpy(minXG, minXCPU, sizeof(float)*testClusterSize, hipMemcpyHostToDevice);
    hipMemcpy(maxXG, maxXCPU, sizeof(float)*testClusterSize, hipMemcpyHostToDevice);
    hipMemcpy(minZG, minZCPU, sizeof(float)*testClusterSize, hipMemcpyHostToDevice);

    findClearPathKernel<<<1, MAX_THREADS>>>(minXG, maxXG, minZG, testClusterSize, leftBearing, rightBearing);
    
    findAngleOffCenterKernel<<<1, MAX_THREADS>>>(minXG, maxXG, minZG, testClusterSize, leftBearing, 0);
    findAngleOffCenterKernel<<<1, MAX_THREADS>>>(minXG, maxXG, minZG, testClusterSize, rightBearing, 1);    
    
    checkStatus(hipGetLastError());
    hipDeviceSynchronize();

    hipFree(minXG);
    hipFree(maxXG);
    hipFree(minZG);
}