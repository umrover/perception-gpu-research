#include "hip/hip_runtime.h"
#include "common.hpp"
#include <float.h>
//Find clear path
//Checks to see if minx and maxX are between the two projected lines
//It not checks to see if the line segment between the two minX and maxX intersect the projected line
//If such points are found launch write them to place in shared memory
//If such poitns are not found then in shared memory space write float_max or float_min
//SyncThreads and perform parallel reduction to find minX and maxX thread values
//launch left and right path kernels using new calculated lines
//Left and right path kernels will perform the same steps except in a particular direction
//Assumes we don't have over 1024 clusters

class compareLine {
public:
    int xIntercept;
    double slope;
    
    __device__ compareLine(double angle_in, int xInt_in) : xIntercept{xInt_in}, 
                        slope{tan(angle_in*PI/180)} {
                            if(slope != 0) {
                                slope = 1/slope;
                            }
                        }

    //Returns 1 if point is right of line, 0 if on, -1 if left of line
    __device__ int operator()(float x, float y) {
        
        //Make sure don't divide by 0
        double xc = xIntercept; //x calculated
        if(slope != 0) {
            xc = y/slope+xIntercept; //Find x value on line with same y value as input point
        }
            
        //Point is right of the line
        if(x > xc) {
            return 1;
        }
        //Point is on the line
        else if (x == xc) {
            return 0;
        }
        //Point is left of the line
        else {
            return -1;
        } 
    }

    //Assumes x1 < x2
    __device__ bool operator()(float x1, float y1, float x2, float y2) {
        if(x1 != x2){
            if(slope != 0){
                float slopeSeg = (y2-y1)/(x2-x1);
                float xIntersect = (-slopeSeg*x1+y1-xIntercept)/(slope-slopeSeg);
                return (xIntersect < x2 && xIntersect > x1);
            }
            //Check if left of line and right of line if slope is undefined
            else if(this->operator()(x1,y1) < 1 && this->operator()(x2,y2) > -1) return true; 
        }
        return false;
        
    }
};

int findNextLargestSquare(int num){
    int exp = log2(num) + 1;
    return pow(2,exp);
}

__global__ void findClearPathKernel(float* minXG, float* maxXG, float* minZG, int numClusters) {
        
    //Declare variables
    const int HALF_ROVER = 5;
    if(threadIdx.x >= numClusters) return;

    __shared__ float maxXS[MAX_THREADS];
    __shared__ float minXS[MAX_THREADS];
    __shared__ float minZS[MAX_THREADS];
    compareLine rightLine(0,HALF_ROVER); //Is there a way to make these objects shared?
    compareLine leftLine(0,-HALF_ROVER);

    float maxX;
    float minX;
    float minZ;

    //Copy over data from global to local
    maxX = maxXG[threadIdx.x];
    minX = minXG[threadIdx.x];
    minZ = minZG[threadIdx.x];

    if(threadIdx.x == 0)
        printf("Vars are copied\n");
    //Check where point is relative to line
    //Since the z values of the min and max x values aren't stored we are going
    //to assume that they are both located at the min z value of the cluster

    //Checks if either of the points is between the two lines
    if((leftLine(maxX, minZ) > -1 && rightLine(maxX, minZ) < 1) || //check if maxX is between right and left
       (leftLine(minX, minZ) > -1 && rightLine(minX, minZ) < 1) || //Check if minX is between right and left
       (leftLine(minX, minZ, maxX, minZ) || rightLine(minZ, minZ, maxX, minZ))) { //check if lines intersect line seg
        maxXS[threadIdx.x] = maxX;
        minXS[threadIdx.x] = minX;
        minZS[threadIdx.x] = minZ;
        printf("Point %d found in path\n", threadIdx.x);
    }
    else {
        maxXS[threadIdx.x] = FLT_MIN;
        minXS[threadIdx.x] = FLT_MAX;
    }

    __syncthreads();

    //Iterate through points to find mins and maxes
    //No need for parallel reduction, since cluster size is relatively small
    
    if(threadIdx.x == 0){
        float minXFinal = FLT_MAX, minZMinXFinal = 0, maxXFinal = FLT_MIN, minZMaxXFinal = 0; 
        for(int i = 0; i < numClusters; ++i ){
            if(maxXS[i] > maxXFinal) {
                maxXFinal= maxXS[i];
                minZMaxXFinal = minZS[i];
                printf("new max found\n");
            }
            if(minXS[i] < minXFinal) {
                minXFinal = minXS[i];
                minZMinXFinal = minZS[i];
                printf("new min found\n");
            }
        }
        float leftBearing;
        leftBearing = (maxXFinal != FLT_MIN) ? 100 : 0; //There is an obstacle in the path

        printf("Max: (%.1f, %.1f)\n", maxXFinal, minZMaxXFinal);
        printf("Min: (%.1f, %.1f)\n", minXFinal, minZMinXFinal);
    }
    
}

void testClearPath() {

    int testClusterSize = 10;
    float* minXG;
    float* maxXG;
    float* minZG;

    hipMalloc(&minXG, sizeof(float)*testClusterSize);
    hipMalloc(&maxXG, sizeof(float)*testClusterSize);
    hipMalloc(&minZG, sizeof(float)*testClusterSize);

    float minXCPU[testClusterSize] = { -6,   -6, 10, 10, 10,   -10, 10, 10, 10, 10};
    float maxXCPU[testClusterSize] = { -6, -5.5, 20, 20, 20,  -9.5, 20, 20, 20, 20};
    float minZCPU[testClusterSize] = { 10,   10, 10, 10, 10,    40, 10, 10, 10, 10};

    hipMemcpy(minXG, minXCPU, sizeof(float)*testClusterSize, hipMemcpyHostToDevice);
    hipMemcpy(maxXG, maxXCPU, sizeof(float)*testClusterSize, hipMemcpyHostToDevice);
    hipMemcpy(minZG, minZCPU, sizeof(float)*testClusterSize, hipMemcpyHostToDevice);

    findClearPathKernel<<<1, MAX_THREADS>>>(minXG, maxXG, minZG, testClusterSize);
    checkStatus(hipGetLastError());
    hipDeviceSynchronize();

    hipFree(minXG);
    hipFree(maxXG);
    hipFree(minZG);
}