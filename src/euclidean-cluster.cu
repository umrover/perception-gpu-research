#include "hip/hip_runtime.h"




/*
This kernel determines the structure of the graph but does not build it
In theory, there is a memory-compute trade off to be made here. This kernel
is not strictly necessary if we allow an upper bound of memory so that each 
point can have the entire dataset amount of neighbors. Perhaps we can 
explore this allocation method instead.
*/
//b: enough, t: each point
__global__ determineGraphStructureKernel(GPU_Cloud_F4 pc, float tolerance, int* listStart) {
    int ptIdx = blockIdx.x * blockDim.x + threadIdx.x;
    sl::float3 pt = pc.data[ptIdx];
    int neighborCount = 0;
    
    //horrible slow way of doing this that is TEMPORARY --> please switch to radix sorted bins
    for(int i = 0; i < pc.size; i++) {
        sl::float3 dvec = (pt - pc.data[i]);
        //this is a neighbor
        if( dvec.norm() < tolerance) {
            neighborCount++;
        }
    }
    listStart[ptIdx] = neighborCount;

    //we must do an exclusive scan using thrust after this kernel
}


/* This kernel builds the graph 
Fairly standard adjacency list structure. 
*/
__global__ buildGraphKernel(GPU_Cloud_F4 pc, float tolerance, int* neighborLists, int* listStart, int* labels, bool* f1, bool* f2) {
    int ptIdx = blockIdx.x * blockDim.x + threadIdx.x;
    sl::float3 pt = pc.data[ptIdx];
    int neighborCount = 0;
    //get the adjacency list for this point
    float* list = neighborLists + listStart[ptIdx]
    
    //horrible slow way of doing this that is TEMPORARY --> please switch to radix sorted bins
    for(int i = 0; i < pc.size; i++) {
        sl::float3 dvec = (pt - pc.data[i]);
        //this is a neighbor
        if( dvec.norm() < tolerance) {
            list[neighborCount] = i;
            neighborCount++;
        }
    }
    
    listStart[ptIdx] = neighborCount;
    labels[ptIdx] = ptIdx;
    f1[ptIdx] = true;
    f2[ptIdx] = false;
}

//this kernel propogates labels, it must be called in a loop
__global__ propogateLabels(GPU_Cloud_F4 pc, int* neighborLists, int* listStart, int* labels, bool* f1, bool* f2, bool* m) {
    int ptIdx = blockIdx.x * blockDim.x + threadIdx.x;

    bool mod = false;

    if(f1[ptIdx]) {
        float* list = neighborLists + listStart[ptIdx]
        int listLen = listStart[ptIdx+1] - listStart[ptIdx];
        f1[ptIdx] = false;
        int myLabel = labels[ptIdx];

        for(int i = 0; i < listLen; i++) {
            int otherLabel = labels[list[i]];
            if(myLabel < otherLabel) { //are these reads actually safe?
                atomicMin(&labels[list[i]], myLabel);
                f2[list[i]] = true;
                *m = true;
            } else {
                myLabel = otherLabel;
                mod = true;
            }
        }
    }

    if(mod) {
        atomicMin(&labels[ptIdx], myLabel);
        f2[ptIdx] = true;
        *m = true
    }
}

//this debug kernel colors points based on their label
__global__ colorClusters(GPU_Cloud_F4 pc, int* labels) {
    int ptIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if(ptIdx == )
}

EuclideanClusterExtractor::EuclideanClusterExtractor(float tolerance, int minSize, float maxSize) 
: tolerance(tolerance), minSize(minSize), maxSize(maxSize) {}

EuclideanClusterExtractor::extractClusters(GPU_Cloud_F4 pc) {

}